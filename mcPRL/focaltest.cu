
#include <hip/hip_runtime.h>
//////#include <iostream>
//////#include <fstream>
//////#include <sstream>
//////
//////#include "CuLayer.h"
//////
//////#include "CuEnvControl.h"
//////#include "LocalOperator.h"
//#include "FocalOperator.h"
//////#include "NeighborhoodSlope.h"
//////#include <time.h>
//////
//////#include "cputest.h"
//////
//////using namespace std;
//////using namespace CuPRL;
//////
//////int main(int argc, char* argv[])
//////{
//////
//////	/*
//////	int testdata[36] = { 1, 1, 1, 1, 1, 1,
//////		1, 3, 3, 2, 1, 10,
//////		1, 1, 3, 2, 2, 2,
//////		1, 2, 2, 2, 2, 2,
//////		1, 1, 1, 2, 2, 2,
//////		1, 1, 1, 1, 1, 2 };
//////
//////
//////	CuLayer<int>testlayer1(testdata, 6, 6);
//////	testlayer1.setCellHeight(1);
//////	testlayer1.setCellWidth(1);
//////	testlayer1.setNoDataValue(10);
//////
//////	printLayer(testlayer1);
//////	*/
//////	//CuEnvControl::setBlockDim(16, 16);
//////
//////	CuLayer<int>testlayer1;
//////	//testlayer1.Read("D:\\cuda\\shikong\\testdata\\M_2002_01_04.tif");
//////	
//////	testlayer1.Read("D:\\cuda\\shikong\\testdata\\nd_dem.tif");
//////	
//////	/*int max = 0;
//////	for (int i = 0; i < testlayer1.getHeight(); i++)
//////	{
//////		for (int j = 0; j < testlayer1.getWidth(); j++)
//////		{
//////			if (testlayer1[i*testlayer1.getWidth() + j] > max&&testlayer1[i*testlayer1[i*testlayer1.getWidth()+j]]!=10000)
//////			{
//////				max = testlayer1[i*testlayer1.getWidth() + j];
//////			}
//////		}
//////	}
//////	cout << "max=" << max << endl;
//////	*/
//////	//cout << testlayer1[0] << endl;
//////
//////
//////
//////	clock_t t1, t2;
//////
//////	t1 = clock();
//////
//////	//NeighborhoodSlope neiSlope;
//////
//////	//CuLayer<double>testgpulayer = cuFocalOperatorFn<int, double, int, SlopeCal>(testlayer1, &neiSlope);
//////
//////
//////	//NeighborhoodRect<int>neiRect(3, 3);
//////	//CuLayer<float>testSumLayer = focalStatisticsMean<int, float, int>(testlayer1, &neiRect, NOUSE, IGNORE);
//////	////CuLayer<int>testSumLayer = testlayer1;
//////	//CuLayer<float>testSumLayer = focalStatisticsMean<int, float, int>(testlayer1, &neiRect, NOUSE, IGNORE);
//////	NeighborhoodSlope neiSlope;
//////CuLayer<double>testgpulayer = cuFocalOperatorFn<int, double, int, SlopeCal>(testlayer1, &neiSlope);
//////	t2 = clock();
//////
//////	cout << t2 - t1 << endl;
//////
//////	t1 = clock();
//////	
//////	cudaDeviceProp device_prop;
//////        /* cudaGetDeviceProperties: ��ȡָ����GPU�豸���������Ϣ */
//////        cudaGetDeviceProperties(&device_prop, 0);
//////		fprintf(stdout, "�豸�϶ദ����������: %d\n", device_prop.multiProcessorCount);
//////	CuLayer<double>testcpulayer = CPUSlopeCal(testlayer1, &neiSlope);
//////
//////	t2 = clock();
//////
//////	cout << t2 - t1 << endl;
//////	/*
//////	if (compareLayer(testcpulayer, testgpulayer) == false)
//////	{
//////		cout << "result error" << endl;
//////	}
//////	else
//////	{
//////		cout << "result right" << endl;
//////	}
//////	*/
//////	/*
//////	int t = 58 + 3251 * testlayer1.getWidth();
//////
//////	for (int i = -1; i < 1; i++)
//////	{
//////		for (int j = -1; j <= 1; j++)
//////		{
//////			cout << testlayer1[t + i*testlayer1.getWidth() + j] << " ";
//////		}
////		cout << endl;
////	}
////	*/
////	testgpulayer.Write("D:\\cuda\\shikong\\testdata\\cuSlope.tif");
////
////	system("pause");
////	return 0;
////}
////
//
//
//
//#include "prpl-dataManager.h"
//#include "aspectTrans.h"
//#include <iostream>
//#include <fstream>
//#include <string>
//#include <cstring>
//#include <cstdio>
//#include"prplcuda.h"
////#include"FocalOperatorDevice.h"
//#include <cstdlib>
//using namespace std;
//
//int main(int argc, char *argv[]) {
//  const string usage("usage: pAspect workspace input-demFilename num-row-subspaces num-col-subspaces task-farming(1/0) io-option(0/1/2/3/4) with-writer(1/0)");
//
//  // Declare a DataManager and initialize MPI
// // bool withWriter = (bool)atoi(argv[argc-1]);
//  bool withWriter = 0;
//  pRPL::DataManager aspDM;
//  if(!aspDM.initMPI(MPI_COMM_WORLD, withWriter)) {
//    cerr << "Error: unable to initialize MPI" << endl;
//    return -1;
//  }
//
//  // Handle arguments
//  //if(argc != 8) {
//  //  if(aspDM.mpiPrc().isMaster()) {
//  //    cout << usage << endl;
//  //  }
//  //  aspDM.finalizeMPI();
//  //  return -1;
//  //}
//  
//  string workspace, demFilename, slopeFilename, aspectFilename;
//  //workspace.assign(argv[1]);
//  //demFilename.assign(workspace + argv[2]);
//  //int nRowSubspcs = atoi(argv[3]);
//  //int nColSubspcs = atoi(argv[4]);
//  //bool taskFarming = (bool)atoi(argv[5]);
//  //int ioOption = atoi(argv[6]);
//  workspace.assign("D:\\cuda\\shikong\\testdata\\");
//  demFilename.assign("D:\\cuda\\shikong\\testdata\\nd_dem.tif");
//  int nRowSubspcs =2;
//  int nColSubspcs = 2;
//  bool taskFarming =0;
//  int ioOption =3;
//  pRPL::ReadingOption readOpt;
//  pRPL::WritingOption writeOpt;
//  string sReadOpt, sWriteOpt;
//  switch(ioOption) {
//    case 0:
//      readOpt = pRPL::CENTDEL_READING;
//      writeOpt = pRPL::NO_WRITING;
//      sReadOpt = "CENTDEL_READING";
//      sWriteOpt = "NO_WRITING";
//      break;
//    case 1:
//      readOpt = pRPL::PARA_READING;
//      writeOpt = pRPL::NO_WRITING;
//      sReadOpt = "PARA_READING";
//      sWriteOpt = "NO_WRITING";
//      break;
//    case 2:
//      readOpt = pRPL::PGT_READING;
//      writeOpt = pRPL::NO_WRITING;
//      sReadOpt = "PGT_READING";
//      sWriteOpt = "NO_WRITING";
//      break;
//    case 3:
//      readOpt = pRPL::CENTDEL_READING;
//      writeOpt = pRPL::CENTDEL_WRITING;
//      sReadOpt = "CENTDEL_READING";
//      sWriteOpt = "CENTDEL_WRITING";
//      break;
//    case 4:
//      readOpt = pRPL::PARA_READING;
//      writeOpt = pRPL::PARADEL_WRITING;
//      sReadOpt = "PARA_READING";
//      sWriteOpt = "PARADEL_WRITING";
//      break;
//    case 5:
//      readOpt = pRPL::PGT_READING;
//      writeOpt = pRPL::PGTDEL_WRITING;
//      sReadOpt = "PGT_READING";
//      sWriteOpt = "PGTDEL_WRITING";
//      break;
//    default:
//      cerr << "Error: invalid ioOption (" << ioOption << ")" << endl;
//      return -1;
//  }
//
//  // Record the start time
//  double timeStart, timeInit, timeCreate, timeRead, timeEnd;
//  aspDM.mpiPrc().sync();
//  if(aspDM.mpiPrc().isMaster()) {
//    //cout << "-------- Start --------" << endl;
//    timeStart = MPI_Wtime();
//  }
//
//  // Add Layers to the DataManager
//  pRPL::Layer *pDemLyr = NULL;
//  if(readOpt == pRPL::PGT_READING) {
//    pDemLyr = aspDM.addLayerByPGTIOL("DEM", demFilename.c_str(), 1, true);
//  }
//  else {
//    pDemLyr = aspDM.addLayerByGDAL("DEM", demFilename.c_str(), 1, true);
//  }
//  const pRPL::SpaceDims &glbDims = *(pDemLyr->glbDims());
//  const pRPL::CellspaceGeoinfo *pGlbGeoinfo = pDemLyr->glbGeoinfo();
//  long tileSize = pDemLyr->tileSize();
//  
//  pRPL::Layer *pSlpLyr = aspDM.addLayer("SLOPE");
// pSlpLyr->initCellspaceInfo(glbDims, typeid(float).name(), sizeof(float), pGlbGeoinfo, tileSize);
//
//  pRPL::Layer *pAspLyr = aspDM.addLayer("ASP");
//  pAspLyr->initCellspaceInfo(glbDims, typeid(float).name(), sizeof(float), pGlbGeoinfo, tileSize);
//  
//  // Add a 3X3 Neighborhood to the DataManager
//  pRPL::Neighborhood* pNbrhd3x3 = aspDM.addNbrhd("Moore3x3");
//  pNbrhd3x3->initMoore(3, 1.0, pRPL::CUSTOM_VIRTUAL_EDGES, 0);
//
//  // Declare a Transition
//  AspectTransition aspTrans;
//  aspTrans.scale(1.0);
//  aspTrans.setNbrhdByName(pNbrhd3x3->name());
//  aspTrans.addInputLyr(pDemLyr->name(), false);
//  aspTrans.addOutputLyr(pSlpLyr->name(), false);
//  aspTrans.addOutputLyr(pAspLyr->name(), true);
//  
//  // Decompose the Layers
//  //cout << aspDM.mpiPrc().id() << ": decomposing Cellspaces...." << endl;
//  if(!aspDM.dcmpLayers(aspTrans, nRowSubspcs, nColSubspcs)) {
//    aspDM.mpiPrc().abort();
//    return -1;
//  }
//
//  aspDM.mpiPrc().sync();
//  if(aspDM.mpiPrc().isMaster()) {
//    timeInit = MPI_Wtime();
//  }
//
//  // Create the output datasets
//  if(writeOpt != pRPL::NO_WRITING) {
//    char nPrcs[10]; sprintf(nPrcs, "%d", aspDM.mpiPrc().nProcesses());
//    slopeFilename.assign(workspace + "slp_" + nPrcs + ".tif");
//    aspectFilename.assign(workspace + "asp_" + nPrcs + ".tif");
//    if(writeOpt == pRPL::PGTDEL_WRITING) {
//      if(!aspDM.createLayerPGTIOL(pSlpLyr->name(), slopeFilename.c_str(), NULL) ||
//         !aspDM.createLayerPGTIOL(pAspLyr->name(), aspectFilename.c_str(), NULL)) {
//        aspDM.mpiPrc().abort();
//        return -1;
//      }
//    }
//    else {
//      if(!aspDM.createLayerGDAL(pSlpLyr->name(), slopeFilename.c_str(), "GTiff", NULL) ||
//         !aspDM.createLayerGDAL(pAspLyr->name(), aspectFilename.c_str(), "GTiff", NULL)) {
//        aspDM.mpiPrc().abort();
//        return -1;
//      }
//    }
//  }
//  pRPL::pCuf pf;
//  pf=&pRPL::Transition::cuFocalOperator<short,float,SlopeMPI>;
//  //  pf=&pRPL::Transition::cuFocalOperator<short,float, float,SlopeMPI>;
// // InitCUDA(aspDM.mpiPrc().id()); 
//  aspDM.mpiPrc().sync();
//  if(aspDM.mpiPrc().isMaster()) {
//    timeCreate = MPI_Wtime();
//  }
//  if(taskFarming) {
//    // Initialize task farming
//    //cout << aspDM.mpiPrc().id() << ": initializing task farm...." << endl;
//    int nSubspcs2Map = withWriter ? 2*(aspDM.mpiPrc().nProcesses()-2) : 2*(aspDM.mpiPrc().nProcesses()-1);
//    if(!aspDM.initTaskFarm(aspTrans, pRPL::CYLC_MAP, nSubspcs2Map, readOpt)) {
//      return -1;
//    }
//
//    aspDM.mpiPrc().sync();
//    if(aspDM.mpiPrc().isMaster()) {
//      timeRead = MPI_Wtime();
//    }
//	
//    // Task farming
//    //cout << aspDM.mpiPrc().id() << ": task farming...." << endl;
//	if(aspDM.evaluate_TF(pRPL::EVAL_ALL, aspTrans, readOpt, writeOpt,pf, false, false) != pRPL::EVAL_SUCCEEDED) {
//      return -1;
//    }
//  }
//  else {
//    //cout << aspDM.mpiPrc().id() << ": initializing static tasking...." << endl;
//    if(!aspDM.initStaticTask(aspTrans, pRPL::CYLC_MAP, readOpt)) {
//      return -1;
//    }
//
//    aspDM.mpiPrc().sync();
//    if(aspDM.mpiPrc().isMaster()) {
//      timeRead = MPI_Wtime();
//    }
//
//    //cout << aspDM.mpiPrc().id() << ": static tasking...." << endl;
//	if(aspDM.evaluate_ST(pRPL::EVAL_ALL, aspTrans, writeOpt, pf,false) != pRPL::EVAL_SUCCEEDED) {
//      return -1;
//    }
//  }
//
//  // Save the output data
//  aspDM.closeDatasets();
//
//  // Record the end time, log computing time
//  aspDM.mpiPrc().sync();
//
//  if(aspDM.mpiPrc().isMaster()) {
//    //cout << "-------- Completed --------" << endl;
//    timeEnd = MPI_Wtime();
//
//    ofstream timeFile;
//    string timeFilename(workspace + "asp_time.csv");
//    timeFile.open(timeFilename.c_str(), ios::app);
//    if(!timeFile) {
//      cerr << "Error: unable to open the time log file" << endl;
//    }
//    timeFile << demFilename << "," \
//        << slopeFilename << "," \
//        << aspectFilename << "," \
//        << aspDM.mpiPrc().nProcesses() << "," \
//        << (withWriter?"WITH_WRITER":"NO_WRITER") << "," \
//        << nRowSubspcs << "," \
//        << nColSubspcs << "," \
//        << (taskFarming?"TF":"ST") << "," \
//        << sReadOpt << "," \
//        << sWriteOpt << "," \
//        << timeInit - timeStart << "," \
//        << timeCreate - timeInit << "," \
//        << timeRead - timeCreate << "," \
//        << timeEnd - timeRead << "," \
//        << timeEnd - timeStart << "," \
//        << endl;
//    timeFile.close();
//
//    //cout << aspDM.ownershipMap() << endl;
//  }
//
//  // Finalize MPI
//  aspDM.finalizeMPI();
//  return 0;
// /* int locId,data[100], tag=8888;  
//    MPI_Status status;  
//    MPI_Init(&argc, &argv) ;  
//    MPI_Comm_rank(MPI_COMM_WORLD, &locId) ;  
//    if(locId == 0) {  
//        MPI_Request events;  
//        MPI_Isend(data, 100, MPI_INT, 1, tag , MPI_COMM_WORLD, &events) ;  
//        MPI_Wait(&events, &status) ;  
//    }  
//    if(locId == 1) {  
//        MPI_Probe(MPI_ANY_SOURCE, tag, MPI_COMM_WORLD, &status);  
//        if (status.MPI_SOURCE==0)  
//            MPI_Recv(data, 100, MPI_INT, 0, tag, MPI_COMM_WORLD, &status) ;  
//    }  
//    MPI_Finalize() ;  
//}*/
// // system("pause");
//}